#include "hip/hip_runtime.h"
﻿
#include <hip/hip_runtime.h>
#include ""
#include <SDL.h>

#include <stdio.h>
#include <iostream>
#include "const.cpp"


float max_Iteration = MAX_ITERATION;
long double  min_reel =  - 2.0;
long double  max_reel =  2.0;
long double  min_imaginary =  - 2.0;
long double  max_imaginary =  2.0;
float centerX = 700;
float centerY = 500;

struct Complex {
    double real;
    double imag;
};

__device__ Complex add(Complex a, Complex b) {
    Complex result;
    result.real = a.real + b.real;
    result.imag = a.imag + b.imag;
    return result;
}

__device__ Complex multiply(Complex a, Complex b) {
    Complex result;
    result.real = a.real * b.real - a.imag * b.imag;
    result.imag = a.real * b.imag + a.imag * b.real;
    return result;
}

__device__ double magnitude(Complex z) {
    return sqrt(z.real * z.real + z.imag * z.imag);
}

__global__ void mandelbrot(uchar3* dev_gpuPixels, int width, int height, long double  min_reel, long double  max_reel, long double  min_imaginary, long double  max_imaginary, int max_Iteration) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= width * height) {
        return; // Out-of-bounds thread
    }
    int x = (blockIdx.x * blockDim.x + threadIdx.x) % width;
    int y = (blockIdx.x * blockDim.x + threadIdx.x) / width;

    double real = min_reel + (x * (max_reel - min_reel) / WIDTH);
    double imag = min_imaginary + (y * (max_imaginary - min_imaginary) / HEIGHT);



    Complex c = { real, imag};
    Complex z = { 0, 0 };

    int iterations = 0;

    while (magnitude(z) < 2.0 && iterations < max_Iteration) {
        Complex zSquared = multiply(z, z);
        z = add(zSquared, c);
        iterations+= 1;
    }

    if (iterations >= max_Iteration) {
        dev_gpuPixels[y * width + x] = { 0, 0, 0 };
    }
    else {
        dev_gpuPixels[y * width + x] = { static_cast<unsigned char>(R_MIN + (R_MAX - R_MIN) * R_REVERSE - (double)(max_Iteration - iterations) / max_Iteration * (R_MAX - R_MIN)),
                                         static_cast<unsigned char>(G_MIN + (G_MAX - G_MIN) * G_REVERSE - (double)(max_Iteration - iterations) / max_Iteration * (G_MAX - G_MIN)),
                                         static_cast<unsigned char>(B_MIN + (B_MAX - B_MIN) * B_REVERSE - (double)(max_Iteration - iterations) / max_Iteration * (B_MAX - B_MIN))};
    }
}


__global__ void juliaSet(uchar3* dev_gpuPixels, int width, int height, Complex number, long double  min_reel, long double  max_reel, long double  min_imaginary, long double  max_imaginary, int max_Iteration) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= width * height) {
        return; // Out-of-bounds thread
    }


    int x = (blockIdx.x * blockDim.x + threadIdx.x) % width;
    int y = (blockIdx.x * blockDim.x + threadIdx.x) / width;

    //dev_gpuPixels[y * width + x] = { 0, 255, 0};
    //return;

    double real = min_reel + (x * (max_reel - min_reel) / WIDTH);
    double imag = min_imaginary + (y * (max_imaginary - min_imaginary) / HEIGHT);
    //printf("\n[%f, %f]", real, imag);
    Complex z = { real, imag };

    int iterations = 0;

    while (magnitude(z) < 2.0 && iterations < max_Iteration) {
        Complex zSquared = multiply(z, z);
        z = add(zSquared, number);
        iterations+= 2;
    }


    if (iterations >= max_Iteration) {
        dev_gpuPixels[y * width + x] = { 0, 0, 0 };
    }
    else {
        //printf("%d", iterations);
        dev_gpuPixels[y * width + x] = { static_cast<unsigned char>(R_MIN + (R_MAX-R_MIN) * R_REVERSE - (double)(max_Iteration - iterations) / max_Iteration * (R_MAX - R_MIN)),
                                         static_cast<unsigned char>(G_MIN + (G_MAX - G_MIN) * G_REVERSE - (double)(max_Iteration - iterations) / max_Iteration * (G_MAX-G_MIN)),
                                         static_cast<unsigned char>(B_MIN + (B_MAX-B_MIN) * B_REVERSE - (double)(max_Iteration - iterations) / max_Iteration * (B_MAX - B_MIN))};
                                         
            
            //static_cast<unsigned char>(25 + (double)(max_Iteration-iterations) / max_Iteration * 25)};
    }
}

__global__ void gaussianBlurInPlace(uchar3* image, int width, int height)
{

    float gaussianKernel5x5[25] = {
        1.0f / 256,  4.0f / 256,  6.0f / 256,  4.0f / 256, 1.0f / 256,
        4.0f / 256, 16.0f / 256, 24.0f / 256, 16.0f / 256, 4.0f / 256,
        6.0f / 256, 24.0f / 256, 36.0f / 256, 24.0f / 256, 6.0f / 256,
        4.0f / 256, 16.0f / 256, 24.0f / 256, 16.0f / 256, 4.0f / 256,
        1.0f / 256,  4.0f / 256,  6.0f / 256,  4.0f / 256, 1.0f / 256
    };
    int kernelSize = 5;
    float* kernel = gaussianKernel5x5;
    int x = (blockIdx.x * blockDim.x + threadIdx.x) % width;
    int y = (blockIdx.x * blockDim.x + threadIdx.x) / width;
    if (x < width && y < height)
    {
        float3 result = make_float3(0.0f, 0.0f, 0.0f);

        for (int i = -kernelSize / 2; i <= kernelSize / 2; i++)
        {
            for (int j = -kernelSize / 2; j <= kernelSize / 2; j++)
            {
                int xOffset = x + i;
                int yOffset = y + j;

                if (xOffset >= 0 && xOffset < width && yOffset >= 0 && yOffset < height)
                {
                    int kernelIndex = (i + kernelSize / 2) * kernelSize + (j + kernelSize / 2);
                    uchar3 pixel = image[yOffset * width + xOffset];
                    result.x += static_cast<float>(pixel.x) * kernel[kernelIndex];
                    result.y += static_cast<float>(pixel.y) * kernel[kernelIndex];
                    result.z += static_cast<float>(pixel.z) * kernel[kernelIndex];
                }
            }
        }

        image[y * width + x] = make_uchar3(static_cast<unsigned char>(result.x),
            static_cast<unsigned char>(result.y),
            static_cast<unsigned char>(result.z));
    }
}


void call_julia(uchar3* dev_gpuPixels, int width, int height, Complex number, long double  min_reel, long double  max_reel, long double  min_imaginary, long double  max_imaginary, int max_Iteration) {
    int nbthread = 1024;
    int numBlocks = (width * height + nbthread - 1) / nbthread;


    juliaSet << <numBlocks, nbthread >> > (dev_gpuPixels, width, height, number, min_reel, max_reel, min_imaginary, max_imaginary, max_Iteration);
    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        printf("\ncall_collisionAoS - fin error: %s\n", hipGetErrorString(cudaStatus));
    }

}

void call_mandelbrot(uchar3* dev_gpuPixels, int width, int height, long double  min_reel, long double  max_reel, long double  min_imaginary, long double  max_imaginary, int max_Iteration) {
    int nbthread = 1024;
    int numBlocks = (width * height + nbthread - 1) / nbthread;


    mandelbrot << <numBlocks, nbthread >> > (dev_gpuPixels, width, height, min_reel, max_reel, min_imaginary, max_imaginary, max_Iteration);
    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        printf("\ncall_collisionAoS - fin error: %s\n", hipGetErrorString(cudaStatus));
    }

}

void call_gaussianBlur(uchar3* dev_gpuPixels, int width, int height) {
    int nbthread = 1024;
    int numBlocks = (width * height + nbthread - 1) / nbthread;


    gaussianBlurInPlace << <numBlocks, nbthread >> > (dev_gpuPixels, WIDTH, HEIGHT);

    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        printf("\ncall_collisionAoS - fin error: %s\n", hipGetErrorString(cudaStatus));
    }

}


int getDisplayFromGpu(uchar3* hostPixels, uchar3* dev_gpuPixels, int width, int height) {

    hipError_t err = hipMemcpy(hostPixels, dev_gpuPixels, width * sizeof(uchar3) * height, hipMemcpyDeviceToHost);

    if (err != hipSuccess) {
        printf("Erreur à l'interieur du blit: %s\n", hipGetErrorString(err));
        return 1;
    }
    return 0;
}

int calculate_max_iter(double zoom_level, int initial_max_iter) {
    // Adjust max_iter based on the zoom level
    return static_cast<int>(initial_max_iter + log2(1.0 + zoom_level) * 20);
}

int main(int argc, char* argv[]) {
    // Initialize SDL
    if (SDL_Init(SDL_INIT_VIDEO) < 0) {
        std::cerr << "SDL initialization failed: " << SDL_GetError() << std::endl;
        return 1;
    }

    // Create a window
    SDL_Window* window = SDL_CreateWindow("SDL Example", SDL_WINDOWPOS_UNDEFINED, SDL_WINDOWPOS_UNDEFINED, WIDTH, HEIGHT, SDL_WINDOW_SHOWN);
    if (!window) {
        std::cerr << "Window creation failed: " << SDL_GetError() << std::endl;
        return 2;
    }

    // Create a renderer
    SDL_Renderer* pRenderer = SDL_CreateRenderer(window, -1, SDL_RENDERER_ACCELERATED);
    if (!pRenderer) {
        std::cerr << "Renderer creation failed: " << SDL_GetError() << std::endl;
        return 3;
    }
    SDL_Texture* pTexture = SDL_CreateTexture(pRenderer, SDL_PIXELFORMAT_RGB24, SDL_TEXTUREACCESS_STREAMING, WIDTH, HEIGHT);

    uchar3* dev_gpuPixels;
    uchar3* hostPixels;

    //Allocation of the Pixels on the gpu
    hipMalloc(&dev_gpuPixels, WIDTH * sizeof(uchar3) * HEIGHT);

    hipHostMalloc(&hostPixels, WIDTH * HEIGHT * sizeof(uchar3));



    //printf("Done");

    bool quit = false;
    SDL_Event events;
    SDL_Point MousePosition;
    bool locked = true;
    Complex number = { 0.5, 0.5};
    while (!quit) {
        while (SDL_PollEvent(&events)) {
            switch (events.type)
            {
            case SDL_QUIT:
                quit = true;
                break;

            case SDL_MOUSEMOTION:
                SDL_GetMouseState(&MousePosition.x, &MousePosition.y);
                break;

            case SDL_MOUSEBUTTONUP:
                locked = !locked;

            case SDL_KEYDOWN :
                SDL_Keycode keyCode = events.key.keysym.sym;
                if (keyCode == SDLK_d) {
                    min_reel += (max_reel - min_reel) / 100;
                    max_reel += (max_reel - min_reel) / 100;
                }
                if (keyCode == SDLK_q) {
                    min_reel -= (max_reel - min_reel) / 100;
                    max_reel -= (max_reel - min_reel) / 100;
                }
                if (keyCode == SDLK_s) {
                    min_imaginary += (max_imaginary - min_imaginary) / 100;
                    max_imaginary += (max_imaginary - min_imaginary) / 100;
                }
                if (keyCode == SDLK_z) {
                    min_imaginary -= (max_imaginary - min_imaginary) / 100;
                    max_imaginary -= (max_imaginary - min_imaginary) / 100;
                }
                if (keyCode == SDLK_a) {
                    min_reel += (max_reel - min_reel) / 100;
                    max_reel -= (max_reel - min_reel) / 100;
                    min_imaginary += (max_imaginary - min_imaginary) / 100;
                    max_imaginary -= (max_imaginary - min_imaginary) / 100;
                }
                if (keyCode == SDLK_e) {
                    min_reel -= (max_reel - min_reel) / 100;
                    max_reel += (max_reel - min_reel) / 100;
                    min_imaginary -= (max_imaginary - min_imaginary) / 100;
                    max_imaginary += (max_imaginary - min_imaginary) / 100;
                }
                if (keyCode == SDLK_r) {
                    min_reel = -2.0;
                    max_reel = 2.0;
                    min_imaginary = -2.0;
                    max_imaginary = 2.0;
                }
            }
        }

        //Complex number = { 0.6, 0.5 };
        if(!locked)
            number = { -2 + (float) MousePosition.x / WIDTH * (2 - -2), -2 + (float) MousePosition.y / HEIGHT * (2 - -2) };

        double zoom_level = 1.0 / (fabs(max_reel - min_reel) < fabs(max_imaginary - min_imaginary) ?
            fabs(max_reel - min_reel) : fabs(max_imaginary - min_imaginary));
        //printf("\n %d", (int)calculate_max_iter(zoom_level, max_Iteration));
        if(MODE==0)
            call_julia(dev_gpuPixels, WIDTH, HEIGHT, number, min_reel, max_reel, min_imaginary, max_imaginary, (int)calculate_max_iter(zoom_level, max_Iteration));
        else if(MODE==1)
            call_mandelbrot(dev_gpuPixels, WIDTH, HEIGHT, min_reel, max_reel, min_imaginary, max_imaginary, (int)calculate_max_iter(zoom_level, max_Iteration));

        //call_gaussianBlur(dev_gpuPixels, WIDTH, HEIGHT);
        getDisplayFromGpu(hostPixels, dev_gpuPixels, WIDTH, HEIGHT);
        SDL_UpdateTexture(pTexture, NULL, hostPixels, WIDTH * sizeof(uchar3));
        SDL_RenderCopy(pRenderer, pTexture, NULL, NULL);
        SDL_RenderPresent(pRenderer);

    }
        SDL_DestroyRenderer(pRenderer);
        SDL_DestroyWindow(window);
        SDL_Quit();

        return 0;

}